#include "hip/hip_runtime.h"
#include "caffe/layers/activation/pooling_layer.hpp"

namespace caffe {


static __global__ void MaxPoolForward(const int nthreads, const float* bottom_data,
    const int num, const int channels, const int height, const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h, const int stride_w, const int pad_h, const int pad_w, 
    float* top_data, float* mask) 
{
  CUDA_KERNEL_LOOP(index, nthreads) 
  {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    float maxval = -FLT_MAX;
    int maxidx = -1;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) 
      for (int w = wstart; w < wend; ++w) 
        if (bottom_data[h * width + w] > maxval) 
        {
          maxidx = h * width + w;
          maxval = bottom_data[maxidx];
        }
      
    
    top_data[index] = maxval;   
    mask[index] = maxidx;    
  }
}


static __global__ void AvePoolForward(const int nthreads, const float* bottom_data,
    const int num, const int channels, const int height, const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h, const int stride_w, const int pad_h, const int pad_w, 
    float* top_data) 
{
  CUDA_KERNEL_LOOP(index, nthreads) 
  {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    float aveval = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) 
      for (int w = wstart; w < wend; ++w) 
        aveval += bottom_data[h * width + w];


    top_data[index] = aveval / pool_size;
  }
}


static __global__ void MaxPoolBackward(const int nthreads, const float* top_diff, const float* mask, 
		const int num, const int channels, const int height, const int width, const int pooled_height, const int pooled_width, 
		const int kernel_h, const int kernel_w, const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    float* bottom_diff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) 
	{
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart =
        (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    int pwstart =
        (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    float gradient = 0;
    int offset = (n * channels + c) * pooled_height * pooled_width;
    top_diff += offset;
  
    mask += offset;
    for (int ph = phstart; ph < phend; ++ph) 
      for (int pw = pwstart; pw < pwend; ++pw) 
        if (mask[ph * pooled_width + pw] == h * width + w) 
          gradient += top_diff[ph * pooled_width + pw];



    
    bottom_diff[index] = gradient;
  }
}


static __global__ void AvePoolBackward(const int nthreads, const float* top_diff,
    const int num, const int channels, const int height, const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    float* bottom_diff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) 
  {
    int w = index % width + pad_w;
    int h = (index / width) % height + pad_h;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int phend = min(h / stride_h + 1, pooled_height);
    int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    int pwend = min(w / stride_w + 1, pooled_width);
    float gradient = 0;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) 
      for (int pw = pwstart; pw < pwend; ++pw) 
      {
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, height + pad_h);
        int wend = min(wstart + kernel_w, width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff[ph * pooled_width + pw] / pool_size;
      }
    bottom_diff[index] = gradient;
  }
}
//--------------------------------------------------------------------------------------

void PoolingLayer::Forward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	
	const float* bottom_data = bottom[0]->gpu_data();
	float* top_data = top[0]->mutable_gpu_data();
	int count = top[0]->count();
	float* mask = NULL;
	if (this->layer_param_.pooling_param().global_pool() == false)
	{
		if (this->layer_param_.pooling_param().pool() == "max") 
		{
			mask = max_idx_.mutable_gpu_data();    
			MaxPoolForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
			(   count, bottom_data,
					num, channels, height, width, pooled_height_, pooled_width_, 
					kernel_size_, kernel_size_, stride_, stride_, pad_, pad_, 
					top_data, mask);  
		}
		else if (this->layer_param_.pooling_param().pool() == "ave") 
		{
			AvePoolForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
			(   count, bottom_data, 
					num, channels, height, width, pooled_height_, pooled_width_, 
					kernel_size_, kernel_size_, stride_, stride_, pad_, pad_, 
					top_data);	
		}	
		else
			LOG(FATAL)<<"unsupported";
	}
	else
	{
		if (this->layer_param_.pooling_param().pool() == "max") 
		{   
			mask = max_idx_.mutable_gpu_data();    
			MaxPoolForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
			(   count, bottom_data,
				  num, channels, height, width, pooled_height_, pooled_width_, 
					height, width, height, width, 0, 0, 
				  top_data, mask);  
		}
		else if (this->layer_param_.pooling_param().pool() == "ave") 
		{
			AvePoolForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
			(   count, bottom_data, 
					num, channels, height, width, pooled_height_, pooled_width_, 
					height, width, height, width, 0, 0, 
					top_data);	

		}
		else
			LOG(FATAL)<<"unsupported";	
	}		
}


void PoolingLayer::Backward_gpu(const vector<Blob*>& top, const vector<Blob*>& bottom) 
{

 	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
		
	 
	const float* top_diff = top[0]->gpu_diff();
	float* bottom_diff = bottom[0]->mutable_gpu_diff();
	const int count = bottom[0]->count();
	caffe_gpu_set(count, float(0.), bottom_diff);
	const float* mask = NULL;
	if (this->layer_param_.pooling_param().global_pool() == false)
	{ 
		if (this->layer_param_.pooling_param().pool() == "max") 
		{
			mask = max_idx_.gpu_data();    
			MaxPoolBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
			(   count, top_diff, mask, 
					num, channels, height, width, pooled_height_, pooled_width_,
					kernel_size_, kernel_size_, stride_, stride_, pad_, pad_, 
					bottom_diff); 
		}
		else if (this->layer_param_.pooling_param().pool() == "ave") 
		{
			AvePoolBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
			(   count, top_diff, 
					num, channels, height, width, pooled_height_, pooled_width_, 
					kernel_size_, kernel_size_, stride_, stride_, pad_, pad_, 
					bottom_diff);
		}
		else
			LOG(FATAL)<<"unsupported";	
	}
	else
	{
		if (this->layer_param_.pooling_param().pool() == "max") 
		{
			mask = max_idx_.gpu_data();    
			MaxPoolBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
			(   count, top_diff, mask, 
					num, channels, height, width, pooled_height_, pooled_width_,
					height, width, height, width, 0, 0, 
					bottom_diff); 
		}
		else if (this->layer_param_.pooling_param().pool() == "ave") 
		{
			AvePoolBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
			(   count, top_diff, 
					num, channels, height, width, pooled_height_, pooled_width_, 
					height, width, height, width, 0, 0, 
					bottom_diff);
		}
		else
			LOG(FATAL)<<"unsupported";	
	}
}
	

void PoolingLayer::SecForward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	
	const float* bottom_data = bottom[0]->gpu_data();
	float* top_data = top[0]->mutable_gpu_data();
	int count = top[0]->count();
	float* mask = NULL;
	
	
	AvePoolForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
			(   count, bottom[0]->gpu_sec_diff(), 
					num, channels, height, width, pooled_height_, pooled_width_, 
					kernel_size_, kernel_size_, stride_, stride_, pad_, pad_, 
					top[0]->mutable_gpu_sec_diff());	
}





}  // namespace caffe
