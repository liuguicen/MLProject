#include "hip/hip_runtime.h"

#include <vector>

#include "caffe/layers/activation/bias_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

static __global__ void forward_kernel(const int count, const int channels, const int spatial_dim, const float* in, const float * b, float* out) 
{
  CUDA_KERNEL_LOOP(i, count) 
  {
  	int c = i / spatial_dim % channels;
  	out[i] = in[i] + b[c];
  }
}


static __global__ void backward_kernel_bias(int num, int channels, int spatial_dim,  const float* top_diff, float* b_diff) 
{
  __shared__ float buffer[CAFFE_CUDA_NUM_THREADS];
  const int tid = threadIdx.x;
  const int c = blockIdx.x;

  // load and accumulate data on each thread
  buffer[tid] = 0;
  for (int i = tid; i < num * spatial_dim; i += blockDim.x) 
  {
    const int index = i / spatial_dim * channels * spatial_dim + c * spatial_dim + i % spatial_dim;
    buffer[tid] += top_diff[index];
  }
  __syncthreads();
  // do tree reduction
  for (int s = blockDim.x / 2; s > 0; s >>= 1) 
  {
    if (tid < s) 
    {
      buffer[tid] += buffer[tid + s];
    }
    __syncthreads();
  }

  // save the result back
  if (tid == 0) 
  {
    b_diff[c] += buffer[0];
  }
}

void BiasLayer::Forward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
	
	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	
	forward_kernel<<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
	(bottom[0]->count(), channels, height*width, bottom[0]->gpu_data(), this->blobs_[0]->gpu_data(), top[0]->mutable_gpu_data());
	
}


void BiasLayer::Backward_gpu(const vector<Blob*>& top, const vector<Blob*>& bottom) 
{
	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	
	caffe_copy(bottom[0]->count(),top[0]->gpu_diff(),bottom[0]->mutable_gpu_diff());
	
	if (this->lr_mult()[0] > 0 && Caffe::frozen_param() == false)
	{		
		backward_kernel_bias<<<channels,CAFFE_CUDA_NUM_THREADS>>>
		(num, channels, height*width,  top[0]->gpu_diff(), this->blobs_[0]->mutable_gpu_diff()); 
	}
}

void BiasLayer::SecForward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
	int num = bottom[0]->num();
	int channels = bottom[0]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
				
	caffe_copy(bottom[0]->count(),bottom[0]->gpu_sec_diff(),top[0]->mutable_gpu_sec_diff());
}

}  // namespace caffe
