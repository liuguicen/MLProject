#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/layers/operator/guided_crf_layer.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

static __global__ void softmax_forward_kernel(const int maxStates,const int nNodes, const float * energy,float * prob)
{
	CUDA_KERNEL_LOOP(n, nNodes)
	{
		for(int s=0;s<maxStates;s++)
			prob[s*nNodes+n] = energy[s*nNodes+n];

		float max_prob = float(-FLT_MAX);
		for(int s=0;s<maxStates;s++)
			max_prob =max(max_prob,prob[s*nNodes+n]);

		for(int s=0;s<maxStates;s++)
			prob[s*nNodes+n] -= max_prob;

		float sum = 0;
		for(int s=0;s<maxStates;s++)
			sum += exp(prob[s*nNodes+n]);

		for(int s=0;s<maxStates;s++)
			prob[s*nNodes+n] = exp(prob[s*nNodes+n]) / sum;
	}
}

static __global__ void softmax_backward_kernel(const int maxStates,const int nNodes, const float * top_diff,const float *prob,float * bottom_diff)
{
	CUDA_KERNEL_LOOP(ind, nNodes*maxStates)
	{
		int n=ind % nNodes;
		int s=ind / nNodes;
		float sum = 0;
		for(int s2=0;s2<maxStates;s2++)
			 sum += top_diff[s2*nNodes+n]*prob[s2*nNodes+n]*(float(s==s2)-prob[s*nNodes+n]);
		bottom_diff[s*nNodes+n] = sum;
	}
}
//--------------------------------------------------------------

static __global__ void vector_product_kernel(const int num,const int channels1,const int channels2, const int spatial_dim,const float * a,const float * b,float *var)//var = a .* b
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*channels1*channels2*num)
	{	
		int n   = ind / spatial_dim / channels1 / channels2;
		int c2  = ind / spatial_dim / channels1 % channels2;
		int c1  = ind / spatial_dim % channels1;
		int s   = ind % spatial_dim;
		
		
		var[ind]=a[(n*channels1+c1)*spatial_dim+s]*b[(n*channels2+c2)*spatial_dim+s];
	}
}

static __global__ void substract_vector_product_kernel(const int num, const int channels1,const int channels2,const int spatial_dim,const float *avg,const float *a,const float *b, float * var)//var = avg - a.*b;
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*channels1*channels2*num)
	{
		int n  = ind / spatial_dim / channels1 / channels2;
		int c2 = ind / spatial_dim / channels1 % channels2;
		int c1 = ind / spatial_dim % channels1;	
		int s  = ind % spatial_dim;
		var[ind]=avg[ind]-a[(n*channels1+c1)*spatial_dim+s]*b[(n*channels2+c2)*spatial_dim+s];
	}
}

static __global__ void inv_var_I_eps_kernel_3(const int num, const int channels, const int spatial_dim, const float eps,float *var_I,float *inv_var_I)
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*num)
	{
		int n = ind / spatial_dim;
		int s = ind % spatial_dim;
		
		for(int c=0;c<channels;c++)
			var_I[(n*channels*channels+(c*channels+c))*spatial_dim+s]=var_I[(n*channels*channels+(c*channels+c))*spatial_dim+s]+eps;

		float det = var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*(var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]-var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s])
				- var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*(var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]-var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s])
				+ var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]*(var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]-var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]);

		inv_var_I[(n*channels*channels+0*channels+0)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+0*channels+1)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+0*channels+2)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]
				-var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]);

		inv_var_I[(n*channels*channels+1*channels+0)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+1*channels+1)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+1*channels+2)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]
				-var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]);

		inv_var_I[(n*channels*channels+2*channels+0)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+2*channels+1)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+2*channels+2)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]
				-var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]);
	}

}

static __global__ void div_sum_kernel_3(const int num, const int channels,const int maxStates,const int spatial_dim,const float *inv_var_I,const float *cov_Ip,
																 float *a)
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*maxStates*num)
	{
		int n = ind / spatial_dim / maxStates;
		int m = ind / spatial_dim % maxStates;
		int s = ind % spatial_dim;
		
		
		a[((n*maxStates+m)*channels+0)*spatial_dim+s] = cov_Ip[((n*maxStates+m)*channels+0)*spatial_dim+s]*inv_var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+1)*spatial_dim+s]*inv_var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+2)*spatial_dim+s]*inv_var_I[(n*channels*channels+0*channels+2)*spatial_dim+s];

		a[((n*maxStates+m)*channels+1)*spatial_dim+s]	= cov_Ip[((n*maxStates+m)*channels+0)*spatial_dim+s]*inv_var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]
																	  + cov_Ip[((n*maxStates+m)*channels+1)*spatial_dim+s]*inv_var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+2)*spatial_dim+s]*inv_var_I[(n*channels*channels+1*channels+2)*spatial_dim+s];

		a[((n*maxStates+m)*channels+2)*spatial_dim+s] = cov_Ip[((n*maxStates+m)*channels+0)*spatial_dim+s]*inv_var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+1)*spatial_dim+s]*inv_var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+2)*spatial_dim+s]*inv_var_I[(n*channels*channels+2*channels+2)*spatial_dim+s];
	}
}

static __global__ void substract_vector_matrix_product_kernel_3(const int num, const int channels,const int maxStates,const int spatial_dim,const float * mean_p,const float * a,const float * mean_I,float *b)//	b = mean_p - mean_I *. a;
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*maxStates*num)
	{
		int n = ind / spatial_dim / maxStates;
		int m = ind / spatial_dim % maxStates;
		int s = ind % spatial_dim;
		
		b[ind] = mean_p[ind]
				   - mean_I[(n*3+0)*spatial_dim+s] * a[((n*maxStates+m)*channels+0)*spatial_dim+s]
				   - mean_I[(n*3+1)*spatial_dim+s] * a[((n*maxStates+m)*channels+1)*spatial_dim+s]
				   - mean_I[(n*3+2)*spatial_dim+s] * a[((n*maxStates+m)*channels+2)*spatial_dim+s];
	}
}

static __global__ void vector_matrix_product_sum_kernel_3(const int num, const int channels,const int maxStates,const int spatial_dim,const float *mean_a,const float *I,const float *mean_b,float *q)// q = I .* mean_a + mean_b;
{

	CUDA_KERNEL_LOOP(ind, spatial_dim*maxStates*num)
	{
		int n = ind / spatial_dim / maxStates;
		int m = ind / spatial_dim % maxStates;
		int s = ind % spatial_dim;
		
		q[ind] = I[(n*3+0)*spatial_dim+s] * mean_a[((n*maxStates+m)*channels+0)*spatial_dim+s]
					 + I[(n*3+1)*spatial_dim+s] * mean_a[((n*maxStates+m)*channels+1)*spatial_dim+s]
				   + I[(n*3+2)*spatial_dim+s] * mean_a[((n*maxStates+m)*channels+2)*spatial_dim+s]
				   + mean_b[ind];
	}

}
//---------------------------------------------

void GuidedCRFLayer::guided_filter_gpu(const int num,const int channels,const int maxStates,const int height,const int width,const float *I,const float * p,float *output_p)
{
	const int spatial_dim=height*width;

	//******************************** prob ************************************
	box_filter_gpu(num,maxStates,height,width,radius,p,mean_p.mutable_gpu_data(),buffer_score.mutable_gpu_data());

	vector_product_kernel<<<CAFFE_GET_BLOCKS(num*channels*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,maxStates,spatial_dim,I,p,Ip);//Ip = I .* p;
	
	box_filter_gpu(num,channels*maxStates,height,width,radius,Ip,mean_Ip,buffer_image_score);


	substract_vector_product_kernel<<<CAFFE_GET_BLOCKS(num*channels*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,maxStates,spatial_dim,mean_Ip,mean_I.gpu_data(),mean_p.gpu_data(), cov_Ip);//cov_Ip = mean_Ip - mean_I .* mean_p;


	inv_var_I_eps_kernel_3<<<CAFFE_GET_BLOCKS(num*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,spatial_dim,eps,var_I.mutable_gpu_data(),inv_var_I.mutable_gpu_data());//inv_var_I=inv(var_I + eps);


	div_sum_kernel_3<<<CAFFE_GET_BLOCKS(num*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,maxStates,spatial_dim,inv_var_I.gpu_data(),cov_Ip,a);//a = cov_Ip ./ inv_var_I;

	box_filter_gpu(num,channels*maxStates,height,width,radius,a,mean_a,buffer_image_score);

	substract_vector_matrix_product_kernel_3<<<CAFFE_GET_BLOCKS(num*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
  (num,channels,maxStates,spatial_dim,mean_p.gpu_data(),a,mean_I.gpu_data(),b.mutable_gpu_data());//	b = mean_p - mean_I .* a;


	box_filter_gpu(num,maxStates,height,width,radius,b.gpu_data(),mean_b.mutable_gpu_data(),buffer_score.mutable_gpu_data());

	vector_matrix_product_sum_kernel_3<<<CAFFE_GET_BLOCKS(num*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,maxStates,spatial_dim,mean_a,I,mean_b.gpu_data(),output_p);// q = I .* mean_a + mean_b;

}


void GuidedCRFLayer::Forward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top)
{
	const float * nodePot = bottom[0]->gpu_data();
	const float * imageData = bottom[1]->gpu_data();


	int num = bottom[0]->num();
	int maxStates = bottom[0]->channels();
	int channels = bottom[1]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	int spatial_dim=height*width;

	int nNodes = num*width *height;
	

	//******************************** image ************************************
	box_filter_gpu(num,channels,height,width,radius,imageData,mean_I.mutable_gpu_data(),buffer_image.mutable_gpu_data());

	vector_product_kernel<<<CAFFE_GET_BLOCKS(num*channels*channels*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,channels,spatial_dim,imageData,imageData,II.mutable_gpu_data());// II = I .* I;

	box_filter_gpu(num,channels*channels,height,width,radius,II.gpu_data(),mean_II.mutable_gpu_data(),buffer_image_image.mutable_gpu_data());

	substract_vector_product_kernel<<<CAFFE_GET_BLOCKS(num*channels*channels*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,channels,spatial_dim,mean_II.gpu_data(),mean_I.gpu_data(),mean_I.gpu_data(), var_I.mutable_gpu_data());//var_I = mean_II - mean_I .* mean_I;
	//-----------------------------------------------------------------------------------


	caffe_copy(tempPot.count(),nodePot,tempPot.mutable_gpu_data());
	for(int iter = 0; iter < maxIter; iter++)
	{
		softmax_forward_kernel<<<CAFFE_GET_BLOCKS(nNodes), CAFFE_CUDA_NUM_THREADS>>>
		(maxStates,nNodes,tempPot.gpu_data(),nodeBel[iter]->mutable_gpu_data());


		guided_filter_gpu(num,channels,maxStates,height,width,imageData,nodeBel[iter]->gpu_data(),filterPot.mutable_gpu_data());
	

		caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, maxStates, nNodes, maxStates,
													(float)1., this->blobs_[0]->gpu_data(), filterPot.gpu_data(),
													(float)0., compatPot.mutable_gpu_data());

		caffe_gpu_add(maxStates*nNodes,float(1),nodePot,alpha,compatPot.gpu_data(),tempPot.mutable_gpu_data());
	}
	caffe_copy(top[0]->count(),tempPot.gpu_data(),top[0]->mutable_gpu_data());
}

void GuidedCRFLayer::Backward_gpu(const vector<Blob*>& top, const vector<Blob*>& bottom)
{
	int num = bottom[0]->num();
	int maxStates = bottom[0]->channels();
	int channels = bottom[1]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	int nNodes = num*width *height;

//----------------------- workspace -------------------------
	myworkspace_[0]->Reshape(num*maxStates,channels,height,width);
	myworkspace_[1]->Reshape(num*maxStates,channels,height,width);
	myworkspace_[2]->Reshape(num*maxStates,channels,height,width);
	
  Ip = myworkspace_[0]->mutable_gpu_data();
  mean_Ip = myworkspace_[0]->mutable_gpu_diff();
  
  cov_Ip = myworkspace_[1]->mutable_gpu_data();
  a = myworkspace_[1]->mutable_gpu_diff();
  
  mean_a = myworkspace_[2]->mutable_gpu_data();
  buffer_image_score  = myworkspace_[2]->mutable_gpu_diff();
//---------------------------------------------------------------- 


	const float *top_diff = top[0]->gpu_diff();
	float * bottom_diff = bottom[0]->mutable_gpu_diff();

	const float * imageData = bottom[1]->gpu_data();
	

	caffe_gpu_set(filterPot.count(),float(0),filterPot.mutable_gpu_diff());
	caffe_gpu_set(compatPot.count(),float(0),compatPot.mutable_gpu_diff());
	caffe_gpu_set(tempPot.count(),float(0),tempPot.mutable_gpu_diff());
	caffe_gpu_set(bottom[0]->count(),float(0),bottom_diff);



	caffe_copy(tempPot.count(),top_diff,tempPot.mutable_gpu_diff());
	
	for(int iter = maxIter-1; iter >= 0; iter--)
	{
		caffe_gpu_add(maxStates*nNodes,alpha,tempPot.gpu_diff(),float(0),compatPot.gpu_diff(),compatPot.mutable_gpu_diff());
		caffe_gpu_add(maxStates*nNodes,float(1) ,tempPot.gpu_diff(),float(1),bottom_diff         ,bottom_diff);


		caffe_gpu_gemm(CblasTrans, CblasNoTrans, maxStates, nNodes, maxStates,
													(float)1., this->blobs_[0]->gpu_data(), compatPot.gpu_diff(),
													(float)0., filterPot.mutable_gpu_diff());

		guided_filter_gpu(num,channels,maxStates,height,width,imageData,filterPot.gpu_diff(),nodeBel[iter]->mutable_gpu_diff());

		softmax_backward_kernel<<<CAFFE_GET_BLOCKS(maxStates*nNodes), CAFFE_CUDA_NUM_THREADS>>>
		(maxStates,nNodes,nodeBel[iter]->gpu_diff(),nodeBel[iter]->gpu_data(),tempPot.mutable_gpu_diff());
	}	
	caffe_gpu_add(tempPot.count(),float(1),tempPot.gpu_diff(),float(1),bottom[0]->gpu_diff(),bottom[0]->mutable_gpu_diff());
}

void GuidedCRFLayer::SecForward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top)
{
}

}  // namespace caffe
