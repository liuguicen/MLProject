#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/func/concat_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
//-------------------------------------------------------

static __global__ void concat_forward(int count,int channels, int i_channels, int cur_channels,int spatial_dim,
																const float *a, float *b)
{
	CUDA_KERNEL_LOOP(i, count)
	{
		int n = i / spatial_dim / i_channels;
		int c = i / spatial_dim % i_channels;
		int s = i % spatial_dim;
		
		b[(n*channels+cur_channels+c)*spatial_dim+s] = a[i];
	}
}

static __global__ void concat_backward(int count,int channels, int i_channels, int cur_channels,int spatial_dim,
																const float * b, float *a)
{
	CUDA_KERNEL_LOOP(i, count)
	{
		int n = i / spatial_dim / i_channels;
		int c = i / spatial_dim % i_channels;
		int s = i % spatial_dim;
		
		a[i] = b[(n*channels+cur_channels+c)*spatial_dim+s];		
	}
}
//----------------------------------------------------

static __global__ void concat2_forward(int count,int channels,int channels0,int channels1,int spatial_dim,
																const float *a0, const float * a1, float *b)
{
	CUDA_KERNEL_LOOP(i, count)
	{
		int n = i / spatial_dim / channels;
		int c = i / spatial_dim % channels;
		int s = i % spatial_dim;
		
		if (c<channels0)
			b[i] = a0[((n*channels0)+c)*spatial_dim+s];
		else 
			b[i] = a1[((n*channels1)+c-channels0)*spatial_dim+s];
		
			
	}
}

static __global__ void concat2_backward(int count,int channels,int channels0,int channels1,int spatial_dim,
																const float * b, float *a0,  float * a1)
{
	CUDA_KERNEL_LOOP(i, count)
	{
		int n = i / spatial_dim / channels;
		int c = i / spatial_dim % channels;
		int s = i % spatial_dim;

		
		if (c<channels0)
			a0[((n*channels0)+c)*spatial_dim+s] = b[i];
		else
			a1[((n*channels1)+c-channels0)*spatial_dim+s] = b[i];
			
	}
}
//----------------------------------------------------

void ConcatLayer::Forward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
	if (bottom.size() == 2)
	{
		int num = bottom[0]->num();
		int channels = top[0]->channels();
		int channels0 = bottom[0]->channels();
		int channels1 = bottom[1]->channels();
		int height = bottom[0]->height();
		int width = bottom[0]->width();

		concat2_forward<<<CAFFE_GET_BLOCKS(top[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
		(top[0]->count(),channels,channels0,channels1,height*width,
					bottom[0]->gpu_data(),bottom[1]->gpu_data(),top[0]->mutable_gpu_data());
	}
	else if (bottom.size() > 2)
	{
		int num = top[0]->num();
		int channels = top[0]->channels();
		int height = top[0]->height();
		int width = top[0]->width();
		
		int cur_channels = 0;
		for (int i =0; i < bottom.size();i++)
		{
			int i_channels = bottom[i]->channels();
			concat_forward<<<CAFFE_GET_BLOCKS(bottom[i]->count()), CAFFE_CUDA_NUM_THREADS>>>
			(bottom[i]->count(),channels,i_channels,cur_channels,height*width,
						bottom[i]->gpu_data(), top[0]->mutable_gpu_data());		
			cur_channels += i_channels;
		}
	}
	else
		LOG(FATAL)<<"wrong bottom.size";
}


void ConcatLayer::Backward_gpu(const vector<Blob*>& top, const vector<Blob*>& bottom) 
{
	if  (bottom.size() == 2)
	{
		int num = bottom[0]->num();
		int channels = top[0]->channels();
		int channels0 = bottom[0]->channels();
		int channels1 = bottom[1]->channels();
		int height = bottom[0]->height();
		int width = bottom[0]->width();

		concat2_backward<<<CAFFE_GET_BLOCKS(top[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
		(top[0]->count(),channels,channels0,channels1,height*width,
					top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff(),bottom[1]->mutable_gpu_diff());		
	}
	else if (bottom.size() > 2)
	{
		int num = top[0]->num();
		int channels = top[0]->channels();
		int height = top[0]->height();
		int width = top[0]->width();
		
		int cur_channels = 0;
		for (int i =0; i < bottom.size();i++)
		{
			int i_channels = bottom[i]->channels();
			concat_backward<<<CAFFE_GET_BLOCKS(bottom[i]->count()), CAFFE_CUDA_NUM_THREADS>>>
			(bottom[i]->count(),channels,i_channels,cur_channels,height*width,
						top[0]->gpu_diff(), bottom[i]->mutable_gpu_diff());		
			cur_channels += i_channels;
		}
	}
	else
		LOG(FATAL)<<"wrong bottom.size";
}

void ConcatLayer::SecForward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
	if (bottom.size() == 2)
	{
		int num = bottom[0]->num();
		int channels = top[0]->channels();
		int channels0 = bottom[0]->channels();
		int channels1 = bottom[1]->channels();
		int height = bottom[0]->height();
		int width = bottom[0]->width();

		concat2_forward<<<CAFFE_GET_BLOCKS(top[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
		(top[0]->count(),channels,channels0,channels1,height*width,
					bottom[0]->gpu_sec_diff(),bottom[1]->gpu_sec_diff(),top[0]->mutable_gpu_sec_diff());
	}
	else if (bottom.size() > 2)
	{
		int num = top[0]->num();
		int channels = top[0]->channels();
		int height = top[0]->height();
		int width = top[0]->width();
		
		int cur_channels = 0;
		for (int i =0; i < bottom.size();i++)
		{
			int i_channels = bottom[i]->channels();
			concat_forward<<<CAFFE_GET_BLOCKS(bottom[i]->count()), CAFFE_CUDA_NUM_THREADS>>>
			(bottom[i]->count(),channels,i_channels,cur_channels,height*width,
						bottom[i]->gpu_sec_diff(), top[0]->mutable_gpu_sec_diff());		
			cur_channels += i_channels;
		}
	}
	else
		LOG(FATAL)<<"wrong bottom.size";
}
}  // namespace caffe
