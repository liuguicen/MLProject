#include "hip/hip_runtime.h"
#include "caffe/solver.hpp"
#include <vector>
#include "caffe/util/format.hpp"

#include "caffe/layers/func/write_image_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/highgui/highgui_c.h>
#define IMAGE_NUM 2
namespace caffe {



void WriteImageLayer::Forward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
	if (bottom.size() == 1)
	{
		if (Solver::iter() % 1000 == 0)
	 	{
			LOG(INFO)<<"---------------writing image-----------------";
			std::vector<float> mean_values_;
			mean_values_.clear();
			mean_values_.resize(3);
			mean_values_[0] = 104.008;
		  mean_values_[1] = 116.669;
		  mean_values_[2] = 122.675;
			int num = bottom[0]->num();
			int channels = bottom[0]->channels();
			int height = bottom[0]->height();
			int width = bottom[0]->width();
			const float * bottom_data = bottom[0]->cpu_data();
			cv::Mat cv_im(height*1,width*1,CV_8UC3);
			//cv::Mat cv_im(height*8,width*8,CV_8UC1);
			for (int i=0;i<1*height;i++)
			{
				unsigned char * data_ptr = cv_im.ptr<uchar>(i);
				for (int j=0;j<1*width;j++)						
				{
					for (int c=0;c<channels;c++)
					{				
						int n = (i/height)*1+(j/width);
						int h = i%height;
						int w = j%width;
						int index = ((n*channels+c)*height+h)*width+w;
						data_ptr[j*channels+c] = min(max(bottom_data[index] + mean_values_[c],float(0)),float(255));		
					}
				}
			}
			std::stringstream ss;
			string filename;
			int gpu_id_;
			CUDA_CHECK(hipGetDevice(&gpu_id_));
			ss<<"generateimage//"<<Solver::iter()<<"GPU"<<gpu_id_<<".jpg";
			ss>>filename;
			cv::imwrite(filename,cv_im);
		}
	}
	else if (bottom.size() == 2)
	{
		if (Solver::iter()%500 == 0 &&  Caffe::gan_type() == "train_gnet")//
 		{
{	
			LOG(INFO)<<"---------------writing image-----------------";
			int num = bottom[0]->num();
			int channels = bottom[0]->channels();
			int height = bottom[0]->height();
			int width = bottom[0]->width();
			const float * bottom_data = bottom[1]->cpu_data();
			cv::Mat cv_im(height*IMAGE_NUM ,width*IMAGE_NUM ,CV_8UC3);
			//cv::Mat cv_im(height*IMAGE_NUM,width*IMAGE_NUM,CV_8UC1);
			for (int i=0;i<IMAGE_NUM *height;i++)
			{
				unsigned char * data_ptr = cv_im.ptr<uchar>(i);
				for (int j=0;j<IMAGE_NUM *width;j++)						
				{
					for (int c=0;c<channels;c++)
					{				
						int n = (i/height)*IMAGE_NUM +(j/width);
						int h = i%height;
						int w = j%width;
						int index = ((n*channels+c)*height+h)*width+w;
						data_ptr[j*channels+c] = min(max((bottom_data[index]*127.5+127.5),float(0)),float(255));
					}
				}
			}
			std::stringstream ss;
			string filename;
			ss<<"generateimage//"<<Solver::iter()<<"_"<<format_int(gpu_id_)<<"_real_.jpg";
			ss>>filename;
			cv::imwrite(filename,cv_im);
}
{		
			LOG(INFO)<<"---------------writing image-----------------";
			int num = bottom[0]->num();
			int channels = bottom[0]->channels();
			int height = bottom[0]->height();
			int width = bottom[0]->width();
			const float * bottom_data = bottom[0]->cpu_data();
			cv::Mat cv_im(height*IMAGE_NUM ,width*IMAGE_NUM ,CV_8UC3);
			//cv::Mat cv_im(height*IMAGE_NUM,width*IMAGE_NUM,CV_8UC1);
			for (int i=0;i<IMAGE_NUM *height;i++)
			{
				unsigned char * data_ptr = cv_im.ptr<uchar>(i);
				for (int j=0;j<IMAGE_NUM *width;j++)						
				{
					for (int c=0;c<channels;c++)
					{				
						int n = (i/height)*IMAGE_NUM +(j/width);
						int h = i%height;
						int w = j%width;
						int index = ((n*channels+c)*height+h)*width+w;
						data_ptr[j*channels+c] = min(max((bottom_data[index]*127.5+127.5),float(0)),float(255));
					}
				}
			}
			std::stringstream ss;
			string filename;
			ss<<"generateimage//"<<Solver::iter()<<"_"<<format_int(gpu_id_)<<"_.jpg";
			ss>>filename;
			cv::imwrite(filename,cv_im);
}
		}
	}
}


void WriteImageLayer::Backward_gpu(const vector<Blob*>& top, const vector<Blob*>& bottom) 
{
	
}

void WriteImageLayer::SecForward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
}

}  // namespace caffe
