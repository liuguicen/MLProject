#include "hip/hip_runtime.h"

#include <vector>

#include "caffe/layers/func/pad_image_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

static __global__ void pad_kernel(int count, int height,int width, int pad, const float *in, float *out)
{
	CUDA_KERNEL_LOOP(i, count)
	{
		int nc = i / (width+pad*2) / (height+pad*2) ;
		int h = i / (width+pad*2) % (height+pad*2);
		int w = i % (width+pad*2);
		if (h < pad)
			h = pad - 1 - h;
		else if (h < pad+height)
			h = h - pad;
		else
			h = height - 1 - (h - (pad+height));
		if (w < pad)
			w = pad - 1 - w;
		else if (w < pad+width)
			w = w - pad;
		else
			w = width - 1 - (w - (pad+width));
		out[i] = in[(nc*height+h)*width+w];			 
	}
}


void PadImageLayer::Forward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
  int height = bottom[0]->height();
  int width = bottom[0]->width();
	
	pad_kernel<<<CAFFE_GET_BLOCKS(top[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
	(top[0]->count(),height,width, pad_, bottom[0]->gpu_data(),top[0]->mutable_gpu_data());
#if 0
FILE *fid = fopen("debug","wb");
fwrite(top[0]->cpu_data(),sizeof(float),top[0]->count(),fid);
fclose(fid);
LOG(FATAL)<<height<<", "<<width;
#endif		
	
}


void PadImageLayer::Backward_gpu(const vector<Blob*>& top, const vector<Blob*>& bottom) 
{
	
}

void PadImageLayer::SecForward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
}

}  // namespace caffe
