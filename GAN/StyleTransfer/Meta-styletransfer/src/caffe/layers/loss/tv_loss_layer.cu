#include "hip/hip_runtime.h"

#include <vector>

#include "caffe/layers/loss/tv_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/format.hpp"

namespace caffe {

static __global__ void tvloss_forward(int count, int channels, int height, int width, const float * data, float *loss)
{
	CUDA_KERNEL_LOOP(i, count)
	{
		int h = i / width % height;
		int w = i % width;
		
		float per_loss = 0;
		if (w < width - 1)
			per_loss += (data[i] - data[i+1]) * (data[i] - data[i+1]);
		
		if (h < height - 1)
			per_loss += (data[i] - data[i+width]) * (data[i] - data[i+width]);
			
		loss[i] = per_loss;	
	
	}
}


static __global__ void tvloss_backward(int count, int channels, int height, int width, const float * data, float *diff)
{
	CUDA_KERNEL_LOOP(i, count)
	{
		int h = i / width % height;
		int w = i % width;
		
		float per_diff = 0;
		if (w < width-1)
			per_diff += 2*(data[i] - data[i+1]);
		if (w > 0)
			per_diff -= 2*(data[i-1] - data[i]);
				
		if (h < height-1)			
			per_diff += 2*(data[i] - data[i+width]);
		if (h > 0)
			per_diff -= 2*(data[i-width] - data[i]);
		
		diff[i] = per_diff;	
	}
}


void TVLossLayer::Forward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
	int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();
  
	tvloss_forward<<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
	(bottom[0]->count(),channels,height,width,bottom[0]->gpu_data(),loss_.mutable_gpu_data());
	
	
	float loss;
	caffe_gpu_asum(bottom[0]->count(),loss_.gpu_data(),&loss);
	top[0]->mutable_cpu_data()[0] = loss / float(bottom[0]->count());
}


void TVLossLayer::Backward_gpu(const vector<Blob*>& top, const vector<Blob*>& bottom) 
{
	int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int height = bottom[0]->height();
  int width = bottom[0]->width();
  
	tvloss_backward<<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>
	(bottom[0]->count(),channels,height,width,bottom[0]->gpu_data(),bottom[0]->mutable_gpu_diff());
	
	float loss_weight_ = top[0]->cpu_diff()[0] / float(bottom[0]->count());
	caffe_gpu_scal(bottom[0]->count(),loss_weight_,bottom[0]->mutable_gpu_diff());
}

void TVLossLayer::SecForward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
}

}  // namespace caffe
