#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/loss/parse_output_layer.hpp"

namespace caffe {



void ParseOutputLayer::Forward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
   const float* bottom_data = bottom[0]->cpu_data();
  float* top_label_data = top[0]->mutable_cpu_data();
  float* top_prob_data = NULL;
  if (out_max_val_) {
    top_prob_data = top[1]->mutable_cpu_data();
  }
  float* max_prob_data = max_prob_.mutable_cpu_data();
  int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int spatial_dim = bottom[0]->height() * bottom[0]->width();
  for (int i = 0; i < num; ++i) {
    caffe_set(spatial_dim, float(0), top_label_data);
    // initialize max value from first plane
    caffe_copy(spatial_dim, bottom_data, max_prob_data);
    for (int j = 1; j < channels; ++j) {
      bottom_data += bottom[0]->offset(0, 1);
      for (int k = 0; k < spatial_dim; ++k) {
        float prob = bottom_data[k];
        if (prob > max_prob_data[k]) {
          max_prob_data[k] = prob;
          top_label_data[k] = j;
        }
      }
    }
    top_label_data += top[0]->offset(1);
    if (out_max_val_) {
      caffe_copy(spatial_dim, max_prob_data, top_prob_data);
      top_prob_data += top[1]->offset(1);
    }
  }
//LOG(INFO)<<" data_height = "<<480;
//FILE *fid = fopen("debug","wb");
//fwrite(top[0]->cpu_data(),sizeof(float), 480 * 480,fid);
//fclose(fid);
//LOG(FATAL)<<" data_height = "<<480;
}


void ParseOutputLayer::Backward_gpu(const vector<Blob*>& top, const vector<Blob*>& bottom) 
{
}

void ParseOutputLayer::SecForward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) 
{
}


}  // namespace caffe
